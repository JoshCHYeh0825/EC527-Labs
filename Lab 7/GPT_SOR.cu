/* 
	nvcc GPT_SOR.cu -o GPT_SOR
*/


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                                       hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

#define N 2048
#define ITERATIONS 2000
#define BLOCK_SIZE 16

void initializeArray1D(float *arr, int len, int seed);

__global__ void sor_kernel(float* A, float* B, int n) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x > 0 && x < n - 1 && y > 0 && y < n - 1) {
        for (int iter = 0; iter < ITERATIONS; iter++) {
            B[y * n + x] = 0.25f * (A[(y - 1) * n + x] + A[(y + 1) * n + x] + A[y * n + (x - 1)] + A[y * n + (x + 1)]);
            __syncthreads();
            A[y * n + x] = B[y * n + x];
            __syncthreads();
        }
    }
}

void sor_host(float* A, float* B, int n) {
    for (int iter = 0; iter < ITERATIONS; iter++) {
        for (int y = 1; y < n - 1; y++) {
            for (int x = 1; x < n - 1; x++) {
                B[y * n + x] = 0.25f * (A[(y - 1) * n + x] + A[(y + 1) * n + x] + A[y * n + (x - 1)] + A[y * n + (x + 1)]);
            }
        }
        memcpy(A, B, n * n * sizeof(float));
    }
}

int main() {

    hipEvent_t start, stop;
    float elapsed_gpu, elapsed_cpu;
    
    float *h_A, *h_B, *h_B_gold;
    float *d_A, *d_B;
    size_t size = N * N * sizeof(float);

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_B_gold = (float*)malloc(size);
    initializeArray1D(h_A, N * N, 2453);
    memcpy(h_B, h_A, size);
    memcpy(h_B_gold, h_A, size);

    CUDA_SAFE_CALL(hipMalloc(&d_A, size));
    CUDA_SAFE_CALL(hipMalloc(&d_B, size));
    CUDA_SAFE_CALL(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(N / BLOCK_SIZE, N / BLOCK_SIZE);
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    sor_kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    CUDA_SAFE_CALL(hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost));
    
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("GPU time: %f ms\n", elapsed_gpu);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    sor_host(h_A, h_B_gold, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_cpu, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("CPU time: %f ms\n", elapsed_cpu);

    int errors = 0;
    for (int i = 0; i < N * N; i++) {
        if (fabs(h_B[i] - h_B_gold[i]) > 1e-6) errors++;
    }
    printf("Mismatches: %d\n", errors);

    CUDA_SAFE_CALL(hipFree(d_A));
    CUDA_SAFE_CALL(hipFree(d_B));
    free(h_A);
    free(h_B);
    free(h_B_gold);
    return 0;
}

void initializeArray1D(float *arr, int len, int seed) {
    int i;
    float randNum;
    srand(seed);
  
    for (i = 0; i < len; i++) {
      randNum = (float) rand();
      arr[i] = randNum;
    }
  }